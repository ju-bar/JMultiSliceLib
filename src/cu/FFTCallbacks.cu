// file FFTCallbacks.cu
// author: Juri Barthel, ju.barthel@fz-juelich.de
// April 21, 2020
//
//
/*
This program is free software : you can redistribute it and/or modify
it under the terms of the GNU General Public License as published by
the Free Software Foundation, either version 3 of the License, or
(at your option) any later version.

This program is distributed in the hope that it will be useful,
but WITHOUT ANY WARRANTY; without even the implied warranty of
MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.See the
GNU General Public License for more details.

You should have received a copy of the GNU General Public License
along with this program.If not, see <https://www.gnu.org/licenses/>
*/
//
// Implementation for cufft callbacks
//

#include <stdio.h>
#include "FFTCallbacks.cuh"



__device__ hipfftComplex MultCLoadCallback(void* in_ptr, size_t index, void* params_ptr, void*)
{
    hipfftComplex* in_data = (hipfftComplex*)in_ptr;
    hipfftComplex in_tmp = in_data[index];
    hipfftComplex in;
    fftCallbackParams* params = (fftCallbackParams*) params_ptr;
    hipfftComplex* load_data = params->load;
    if (load_data != NULL) {
        hipfftComplex dat = params->load[index];
        in.x = in_tmp.x * dat.x - in_tmp.y * dat.y;
        in.y = in_tmp.x * dat.y + in_tmp.y * dat.x;
    }
    else {
        in = in_tmp;
    }
    return in;
}

__device__ void MultCStoreCallback(void* out_ptr, size_t index, hipfftComplex in, void* params_ptr, void*)
{
    fftCallbackParams* params = (fftCallbackParams*)params_ptr;
    hipfftComplex out;
    hipfftComplex* store_data = params->store;
    // do propagator multiplication
    if (store_data != NULL) {
        hipfftComplex store = store_data[index];
        out.x = in.x * store.x - in.y * store.y;
        out.y = in.x * store.y + in.y * store.x;
    }
    else {
        out = in;
    }
    hipfftComplex* out_data = (hipfftComplex*)out_ptr;
    out_data[index] = out;
}

